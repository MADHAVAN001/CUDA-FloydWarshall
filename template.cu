#include "hip/hip_runtime.h"
/* Implementation of Floyd-Warshall Algorithm in CUDA
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include<sys/time.h>
#include "MatUtil.h"
#include "MatUtil.c"


// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
void shortestPath(int argc, char **argv);
extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(int *result, int N)
{
    int j,k;
	int Row = blockIdx.x*blockDim.x+threadIdx.x;
	
	for(k = 0;k<N;k++)
	{
		for(j = 0;j<N;j++)
		{
			int l = Row*N+j;
			int m = Row*N+k;
			int n = k*N+j;
			if(result[m] == -1 || result[n] == -1)
				continue;
			else
				if(result[l] == -1)
				result[l] = result[m]+result[n];
			else
				result[l] =  (result[l] < result[m] + result[n])? result[l]:(result[m]+result[n]);
			__syncthreads();
		}
	}
}




////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    shortestPath(argc, argv);
}


void shortestPath(int argc, char **argv)
{
	//bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
    	int devID = findCudaDevice(argc, (const char **)argv);

    	StopWatchInterface *timer = 0;
    	sdkCreateTimer(&timer);
    	sdkStartTimer(&timer);
	
	printf("Starting to generate random matrix for input...\n");	
	fflush(stdin);
	//struct timeval tv1,tv2,tv3,tv4;
	//generate a random matrix.
	size_t N = atoi(argv[1]);
	int *mat = (int*)malloc(sizeof(int)*N*N);
	GenMatrix(mat, N);
	printf("Finished generating the test data....\n");	


	int *result = (int*)malloc(sizeof(int)*N*N);
	//compute the reference result.
	int *ref = (int*)malloc(sizeof(int)*N*N);
	memcpy(ref, mat, sizeof(int)*N*N);
	//gettimeofday(&tv1,NULL);
	ST_APSP(ref, N);
	//gettimeofday(&tv2,NULL);
	//fprintf(f,"%ld,", (tv2.tv_sec -tv1.tv_sec)*1000000+tv2.tv_usec-tv1.tv_usec);
	
	unsigned int mem_size = sizeof(int) * N*N;
	unsigned int num_threads = N;
	
	printf("Finished generating all the matrices\n");	
	int *d_mat;
	
	//Allocate memory for data matrix
	checkCudaErrors(hipMalloc((void **) &d_mat, mem_size));
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(d_mat, mat, mem_size,
                               hipMemcpyHostToDevice));
	
	
	
	//Allocate memory for reference matrix in the device
	int *d_ref;
	
	//Allocate memory for data matrix
	checkCudaErrors(hipMalloc((void **) &d_ref, mem_size));
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(d_ref, ref, mem_size,
                               hipMemcpyHostToDevice));
	
	//Allocate memory for the result
	int *d_result;
	
	//Allocate memory for data matrix
	checkCudaErrors(hipMalloc((void **) &d_result, mem_size));
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(d_result, mat, mem_size,
                               hipMemcpyHostToDevice));
	
	 
	// setup execution parameters
    dim3  grid(1);
    dim3  threads(num_threads, 1, 1);
	
	// execute the kernel
    testKernel<<< grid, threads >>>(d_result, N);
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(result,d_result, mem_size,
                               hipMemcpyDeviceToHost));
	
	//compare your result with reference result
	if(CmpArray(result, ref, N*N))
		printf("Your result is correct.\n");
	else
		printf("Your result is wrong.\n");
		
	checkCudaErrors(hipFree(d_result));
	free(result);
	checkCudaErrors(hipFree(d_ref));
	free(ref);
	free(mat);
}
