#include "hip/hip_runtime.h"
/* Implementation of Floyd-Warshall Algorithm in CUDA
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include<sys/time.h>
#include "MatUtil.h"
#include "MatUtil.c"


// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
void shortestPath(int argc, char **argv);
extern "C"
void computeGold(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(int *result, int N, int k)
{
	//this part needs to be changed
	const int num = 100;
	__shared__ int ds_element_ik;
	__shared__ int ds_Column_k[100];
	__shared__ int ds_row_j[100];
	
	
    int j;
	int Row = blockIdx.x*blockDim.x+threadIdx.x;
	
	ds_element_ik = result[Row*N+k];
	
	for(int tmp = 0;tmp<N;tmp++)
	{
		ds_Column_k[tmp] = result[k*N+tmp];
		ds_row_j[tmp] = result[Row*N+tmp];
	}
	
		for(j = 0;j<N;j++)
		{
			if(ds_element_ik == -1 || ds_Column_k[j] == -1)
				continue;
			else
				if(ds_row_j[j] == -1)
				{
				result[Row*N+j] = ds_element_ik+ds_Column_k[j];
				}
			else
				result[Row*N+j] =  (ds_row_j[j] < ds_element_ik+ds_Column_k[j])? ds_row_j[j]:(ds_element_ik+ds_Column_k[j]);
			
		}

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    shortestPath(argc, argv);
}


void shortestPath(int argc, char **argv)
{
	//bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
    	int devID = findCudaDevice(argc, (const char **)argv);

    	StopWatchInterface *timer = 0;
    	sdkCreateTimer(&timer);
    	sdkStartTimer(&timer);
	
	printf("Starting to generate random matrix for input...\n");	
	fflush(stdin);
	//struct timeval tv1,tv2,tv3,tv4;
	//generate a random matrix.
	size_t N = atoi(argv[1]);
	int *mat = (int*)malloc(sizeof(int)*N*N);
	GenMatrix(mat, N);
	printf("Finished generating the test data....\n");	


	int *result = (int*)malloc(sizeof(int)*N*N);
	//compute the reference result.
	int *ref = (int*)malloc(sizeof(int)*N*N);
	memcpy(ref, mat, sizeof(int)*N*N);
	//gettimeofday(&tv1,NULL);
	ST_APSP(ref, N);
	//gettimeofday(&tv2,NULL);
	//fprintf(f,"%ld,", (tv2.tv_sec -tv1.tv_sec)*1000000+tv2.tv_usec-tv1.tv_usec);
	
	unsigned int mem_size = sizeof(int) * N*N;
	unsigned int num_threads = N;
	
	printf("Finished generating all the matrices\n");	
	int *d_mat;
	
	//Allocate memory for data matrix
	checkCudaErrors(hipMalloc((void **) &d_mat, mem_size));
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(d_mat, mat, mem_size,
                               hipMemcpyHostToDevice));
	
	
	
	//Allocate memory for reference matrix in the device
	int *d_ref;
	
	//Allocate memory for data matrix
	checkCudaErrors(hipMalloc((void **) &d_ref, mem_size));
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(d_ref, ref, mem_size,
                               hipMemcpyHostToDevice));
	
	//Allocate memory for the result
	int *d_result;
	
	//Allocate memory for data matrix
	checkCudaErrors(hipMalloc((void **) &d_result, mem_size));
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(d_result, mat, mem_size,
                               hipMemcpyHostToDevice));
	
	 
	// setup execution parameters
    dim3  grid(1);
    dim3  threads(num_threads, 1, 1);
	
	int k = 0;
	for(k = 0;k<N;k++)
	{
	// execute the kernel
    testKernel<<< grid, threads >>>(d_result, N,k);
	}
	
	// copy host memory to device
    checkCudaErrors(hipMemcpy(result,d_result, mem_size,
                               hipMemcpyDeviceToHost));
	
	//compare your result with reference result
	if(CmpArray(result, ref, N*N))
		printf("Your result is correct.\n");
	else
		printf("Your result is wrong.\n");
		
	checkCudaErrors(hipFree(d_result));
	free(result);
	checkCudaErrors(hipFree(d_ref));
	free(ref);
	free(mat);
}
