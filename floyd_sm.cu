#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

extern "C" {
#include "MatUtil.h"
}

#define TILE_WIDTH 32
#define TILE_HEIGHT 32


__device__
int Min(int a, int b) { return a < b ? a : b; }

__global__
void SharedMemoryFloydWarshall(int* mat, int k, int N) {
    __shared__ int dist_i_k[TILE_HEIGHT];
    __shared__ int dist_k_j[TILE_WIDTH];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < N && j < N) {
        int dist_i_j = mat[i*N + j];
        if (i % TILE_HEIGHT == 0) {
            dist_k_j[j % TILE_WIDTH] = mat[k*N + j];
        }
        if (j % TILE_WIDTH == 0) {
            dist_i_k[i % TILE_HEIGHT] = mat[i*N + k];
        }
        __syncthreads();
        if (dist_i_k[i % TILE_HEIGHT] != -1 && dist_k_j[j % TILE_WIDTH] != -1) {
            int new_dist = dist_i_k[i % TILE_HEIGHT] + dist_k_j[j % TILE_WIDTH];
            if (dist_i_j != -1) {
                new_dist = Min(new_dist, dist_i_j);
            }
            mat[i*N + j] = new_dist;
        }
    }
}

void SharedMemoryFloydWarshallDriver(int* mat, int N, dim3 thread_per_block) {
    int* cuda_mat;
    int size = sizeof(int) * N * N;
    hipMalloc((void**) &cuda_mat, size);
    hipMemcpy(cuda_mat, mat, size, hipMemcpyHostToDevice);
    dim3 num_block(ceil(1.0*N/thread_per_block.x),
                   ceil(1.0*N/thread_per_block.y));
    for (int k = 0; k < N; ++k) {
        SharedMemoryFloydWarshall<<<num_block, thread_per_block>>>(cuda_mat, k, N);
    }
    hipMemcpy(mat, cuda_mat, size, hipMemcpyDeviceToHost);
    hipFree(cuda_mat);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    if(argc != 3) {
        printf("Usage: test {N} {run_sequential_check: 'T' or 'F'}\n");
        exit(-1);
    }
    char run_sequential_check = argv[2][0];
    dim3 thread_per_block(TILE_HEIGHT, TILE_WIDTH);
    //generate a random matrix.
    size_t N = atoi(argv[1]);
    int *mat = (int*)malloc(sizeof(int)*N*N);
    GenMatrix(mat, N);

    //compute your results
    int *result = (int*)malloc(sizeof(int)*N*N);
    memcpy(result, mat, sizeof(int)*N*N);
    //replace by parallel algorithm
    SharedMemoryFloydWarshallDriver(result, N, thread_per_block);
    
    //compare your result with reference result
    if (run_sequential_check == 'T') {
        int *ref = (int*)malloc(sizeof(int)*N*N);
        memcpy(ref, mat, sizeof(int)*N*N);
        ST_APSP(ref, N);
        if(CmpArray(result, ref, N*N))
            printf("Your result is correct.\n");
        else
            printf("Your result is wrong.\n");
#ifdef PRINT_MATRIX
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                printf("%d ", ref[i*N+j]);
            }
            printf("\n");
        }
#endif
    }

#ifdef PRINT_MATRIX
    printf("==RESULT==\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", result[i*N+j]);
        }
        printf("\n");
    }
#endif
}
