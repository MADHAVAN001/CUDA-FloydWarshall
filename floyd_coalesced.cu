#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

extern "C" {
#include "MatUtil.h"
}

__device__
int Min(int a, int b) { return a < b ? a : b; }

__global__
void NaiveFloydWarshall(int* mat, int k, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N*N) {
        int i = idx/N;
        int j = idx - i*N;
        if (mat[i*N + k] != -1 && mat[k*N + j] != -1) {
            if (mat[idx] == -1) {
                mat[idx] = mat[i*N + k] + mat[k*N +j];
            } else {
                mat[idx] = Min(mat[i*N + k] + mat[k*N + j], mat[idx]);
            }
        }
    }
}

void NaiveFloydWarshallDriver(int* mat, int N, int thread_per_block) {
    int* cuda_mat;
    int size = sizeof(int) * N * N;
    hipMalloc((void**) &cuda_mat, size);
    hipMemcpy(cuda_mat, mat, size, hipMemcpyHostToDevice);
    int num_block = ceil(1.0*N*N/(thread_per_block));
    for (int k = 0; k < N; ++k) {
        NaiveFloydWarshall<<<num_block, (thread_per_block)>>>(cuda_mat, k, N);
    }
    hipMemcpy(mat, cuda_mat, size, hipMemcpyDeviceToHost);
    hipFree(cuda_mat);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv)
{
    if(argc != 4) {
        printf("Usage: test {N} {run_sequential_check: 'T' or 'F'} {thread_per_block}\n");
        exit(-1);
    }
    char run_sequential_check = argv[2][0];
    int thread_per_block = atoi(argv[3]);
    //generate a random matrix.
    size_t N = atoi(argv[1]);
    int *mat = (int*)malloc(sizeof(int)*N*N);
    GenMatrix(mat, N);

    //compute your results
    int *result = (int*)malloc(sizeof(int)*N*N);
    memcpy(result, mat, sizeof(int)*N*N);
    //replace by parallel algorithm
    NaiveFloydWarshallDriver(result, N, thread_per_block);
    
    //compare your result with reference result
    if (run_sequential_check == 'T') {
        int *ref = (int*)malloc(sizeof(int)*N*N);
        memcpy(ref, mat, sizeof(int)*N*N);
        ST_APSP(ref, N);
        if(CmpArray(result, ref, N*N))
            printf("Your result is correct.\n");
        else
            printf("Your result is wrong.\n");
    }
}
